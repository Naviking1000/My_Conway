#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "pluh.cuh"

__global__ void GOL(bool* prevFrame, bool* currFrame, int dim)
{
    size_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index > dim*dim) return;
    int neighbors = 0;
    size_t kernel[] = {
        index - (dim + 1), index - dim, index - dim + 1, 
        index - 1, index + 1,
        index + dim - 1, index + dim, index + dim + 1
    };
    for (int i = 0; i < 8; i++)
    {
        if (kernel[i] <= dim*dim)
        {
            if (prevFrame[kernel[i]] == 1)
            {
                neighbors++;
            }
        }
    }
    // if (neighbors < 2)
    // {
    //     currFrame[index] = 0;
    // }
    // if (neighbors == 3)
    // {
    //     currFrame[index] = 1;
    // }
    // if (neighbors > 3)
    // {
    //     currFrame[index] = 0;
    // }
    if (neighbors % 2 == 1)
    {
        currFrame[index] = 1;
    }
    else
    {
        currFrame[index] = 0;
    }
}

__global__ void SyncPrevFrame(bool* prevFrame, bool* currFrame, int dim)
{
    size_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index > dim*dim) return;
    prevFrame[index] = currFrame[index];
}

__global__ void UpdateBuffer(bool* currFrame, uint8_t* buf, int dim, bool paused)
{
    size_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index > dim*dim) return;
    size_t bufIndex = index * 4;
    if (currFrame[index] == 1)
    {
        buf[bufIndex] = 0xff;
        buf[bufIndex + 1] = 0xff;
        buf[bufIndex + 2] = 0xff;
    }
    else
    {
        uint8_t blue = paused ? 0xff : 0x00;
        buf[bufIndex] = 0x00;
        buf[bufIndex + 1] = 0x00;
        buf[bufIndex + 2] = blue;
    }
}

int Pluh::SigmaBoy()
{
    imageSize = width * width * 4;
    hipMallocManaged(&buf, imageSize);
    hipMallocManaged(&prevFrame, width*width);
    hipMallocManaged(&currFrame, width*width);
    for (int i = 0; i < imageSize; i++)
    {
        buf[i] = 0xff;
    }
    for (int i = 0; i < width*width; i++)
    {
        int pixel = rand() % 2;
        currFrame[i] = pixel == 1;
        prevFrame[i] = pixel == 1;
    }

    return 0;
}

void Pluh::Simulate()
{
    int size = width*width;
    int blockSize = 256;
    int blocks = (size + blockSize - 1) / blockSize;

    SyncPrevFrame<<<blocks, blockSize>>>(prevFrame, currFrame, width);
    GOL<<<blocks, blockSize>>>(prevFrame, currFrame, width);
}

void Pluh::DrawPixel(int x, int y, bool value)
{
    size_t pixelIndex = y * width + x;

    if (pixelIndex > width*width) return;

    int size = width*width;
    int blockSize = 256;
    int blocks = (size + blockSize - 1) / blockSize;

    currFrame[pixelIndex] = value;
    prevFrame[pixelIndex] = value;
    hipDeviceSynchronize();
}

void Pluh::ClearGrid()
{
    for (int i = 0; i < width*width; i++)
    {
        prevFrame[i] = 0;
        currFrame[i] = 0;
    }
    hipDeviceSynchronize();
}

uint8_t* Pluh::GetBuf(bool paused)
{
    int size = width*width;
    int blockSize = 256;
    int blocks = (size + blockSize - 1) / blockSize;

    UpdateBuffer<<<blocks, blockSize>>>(currFrame, buf, width, paused);
    hipDeviceSynchronize();
    return buf;
}

size_t Pluh::GetBufSize()
{
    return imageSize;
}

void Pluh::End()
{
    hipFree(buf);
    hipFree(prevFrame);
    hipFree(currFrame);
}

Pluh::Pluh(int width)
{
    this->width = width;
}